#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_runtime.h>

#include "config.h"
#define WPB 8

using namespace nvcuda;

//////////////////////
/// SPMM forward (GCN, GraphSAGE)
//////////////////////
__global__ void QGTC_forward_cuda_kernel(
	const int * __restrict__ nodePointer,		// node pointer.
	const int *__restrict__ edgeList,			// edge list.
);

////////////////////////////////////////////
//
// SPMM Foward Pass  (GCN, GraphSAGE)
//
////////////////////////////////////////////
std::vector<torch::Tensor> QGTC_forward_cuda(
    torch::Tensor A_mat,
    torch::Tensor X_mat,
    torch::Tensor W_1,
    torch::Tensor W_2,
    const int w_bit,
    const int act_bit
) 
{
    auto output = torch::zeros_like(input);

    // quant_A from float to 1-w_bit.
    // quant X from float to act_bit.
    // quant W_1 from float to w_bit.
    // quant W_2 from float to w_bit.

    // dim3 grid(num_row_windows, 1, 1);
    // dim3 block(WARP_SIZE, WARPperBlock, 1);

    // const int dimTileNum = (embedding_dim + BLK_H - 1) / BLK_H;
	// const int dynamic_shared_size = dimTileNum*BLK_W * BLK_H * sizeof(float); // dynamic shared memory.

    // spmm_forward_cuda_kernel<<<grid, block, dynamic_shared_size>>>(
    //                                                                 nodePointer.data<int>(), 
    //                                                                 edgeList.data<int>(),
    //                                                                 blockPartition.data<int>(), 
    //                                                                 edgeToColumn.data<int>(), 
    //                                                                 edgeToRow.data<int>(), 
    //                                                                 num_nodes,
    //                                                                 num_edges,
    //                                                                 embedding_dim,
    //                                                                 input.data<float>(), 
    //                                                                 output.data<float>()
    //                                                             );

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    return {output};
}

__kernel__ void QGTC_layer_input(){

}

__kernel__ void QGTC_layer_hidden(){

}

__kernel__ void QGTC_layer_output(){

}

// input: bit_A_mat, bit_X_mat, bit_W_mat, bit_hidden
// hidden: bit_A_mat, bit_X_mat, bit_W_mat, bit_hidden
// output: bit_A_mat, bit_X_mat, bit_W_mat, bit_output

__global__ void QGTC_forward_cuda_kernel(
    InputParam layer1,
    OutputParam layer2
){

    QGTC_layer_input(layer1);
    grid.sync();

    QGTC_layer_output(layer2);
    grid.sync();
}